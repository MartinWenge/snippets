// minimal example on using host device functions
// compile with:
// nvcc -o hostDev main.cu


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

// CUDA global constants
__constant__ __device__ int32_t dconst = 5;
const int32_t hconst = 10;

__host__ __device__ void foo(){
    #ifdef __CUDA_ARCH__
        printf("%d \n", dconst);
    #else
        printf("%d \n", hconst);
    #endif
}

__global__ void fooKernel(){
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
    foo();
}

int main()
{
    // Kernel invocation with N threads

    std::cout << "run the cuda kernel" << std::endl;
    fooKernel<<<1, 4>>>();
    hipDeviceSynchronize();

    std::cout << "run the host function" << std::endl;
    foo();

    return 0;

}